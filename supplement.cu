#include "hip/hip_runtime.h"
#include <bitset>
#include "supplement.h"
/*
TODO: 
- Get data off the wire/file and put into a given array/vector
- Extract the date from the packets, send extracted packet data to an array/vector 
- Then send data to the GPU to be hashed out by a given kernel
- 
*/

//-----------------------------------------------------------------------------
//CUDA kernel cuda
//----------------------------------------------------------------------------
__device__ void lock(int* mutex){
  while(atomicCAS(mutex, 0, 1) != 0){

  }
}

__device__ void unlock(int*mutex){
  atomicExch(mutex,0);
}

__global__ void
vectorAdd(const float* A, const float* B, float* C, int numElem){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i< numElem){
    C[i] = A[i] + B[i];
  }
}

__global__ void
DHashData(u_char* pkts, size_t ipitch, u_char* output, size_t opitch, size_t dataLength, int numElem){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  u_char*term = pkts + idx*ipitch;
  u_char*dest = output + idx*opitch;
  if(idx < numElem){
	  sha1Device(term, dataLength, dest);
  }
}

__global__ void
hashSeed(u_int seed, u_int series, u_char* output, size_t opitch, size_t dataLength, int numElem){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  u_int temp = (seed + (idx + (series * numElem)));
  u_char*term = (u_char*) &temp;
  u_char*dest = output + idx*opitch;
  if(idx < numElem){
	  sha1Device(term, dataLength, dest);
  }
}

__global__ void
singleSHA1(u_char* src, int size, u_char* dst){
  sha1Device(src, size, dst);
}

__global__ void
findMatch(u_char* hashed, size_t hashedPitch, int hashedSize, u_char* pool, size_t poolPitch, int poolSize, int hashLength, u_int* result){
}

__global__ void
findMatchV2(u_char* dataPoints, size_t dataPitch, u_int dataSize, u_int* identifiers, u_int identSize, u_int bValue,  u_int maxIDs, u_int throwAway, u_int* result){
  int idx = blockIdx.x * threadIdx.x + blockDim.x;
  if(idx<((dataSize*(maxIDs - throwAway)))){
    int mdPosition = idx / (maxIDs - throwAway);
    u_char* currMD = dataPoints + (mdPosition * MD_LENGTH);
    int identityIndex = idx % (maxIDs -throwAway + 1);
    int tempID = getNthIdentifier((u_int*)currMD, identityIndex, bValue, MD_LENGTH);
    int isS = isSet((u_int*) identifiers, tempID);
//    printf("mdPosition:, %d, identityIndex:, %d, tempID;, %d, isS:, %d\n", mdPosition, identityIndex, tempID, isS);
//    result[tempID] += isSet((u_int*)identifiers, tempID);
    if(isS == 0){
      result[mdPosition] = 1;
    }
  }
}

__global__ void
byteToInt(int* out, u_char* in, int n){
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  for(; idx < n; idx += gridDim.x * blockDim.x){
    out[idx] = in[idx];
  }
}

//The Host does not need this function however for testing purposes, its avaiable to the host
__host__ __device__ int
memcmpDevice(const void* s1, const void* s2, size_t n){
  // ret < 0 if s1 is greater, 0 if equal, ret > 0 if s2 is greater
  int ret = 0;
  u_char* t1 = (u_char*) s1;
  u_char* t2 = (u_char*) s2;
  for(int i = 0; i < n; i++){
   if( (t1 && !t2) || (*t1 > *t2 )) {
      return -1;
    }
    if( (!t1 && t2) || (*t1 < *t2)) {
      return 1;
    }
    t1++;
    t2++;
  }
  return ret;
}

__host__ __device__ void
memcpyDevice(void* dest, const void* src, size_t n){
  u_char* s1 = (u_char*)dest;
  u_char* s2 = (u_char*)src;
  for(int i = 0; i < n; i++){
    *s1 = *s2;
    s1++;
    s2++;
  }
}

extern "C" bool
handleData(u_char* dataPool, int dataLength, int dataPoolSize, 
  u_char* secretPool, int sPoolSize, u_int* ret, struct btimes* times)
{
  bool chk = true;
  u_char* remoteData;
  u_char* remoteHashed;
  u_char* remotePool;
  u_int* remoteResult;
//  u_int* remoteScratch;
  size_t remoteDataPitch = 0;
  size_t remoteHashedPitch = 0;
  size_t remotePoolPitch = 0;
  StopWatchInterface* timer = NULL;
  hipError_t error;
  sdkCreateTimer(&timer);
  
  //mallocs
  if(times) sdkStartTimer(&timer); 
  checkCudaErrors(hipMallocPitch((void**)&remoteData, &remoteDataPitch,
		  	  	  	  dataLength*sizeof(u_char), dataPoolSize));
  checkCudaErrors(hipMallocPitch((void**)&remotePool, &remotePoolPitch,
		  	  	  	  MD_LENGTH*sizeof(u_char), sPoolSize));
  checkCudaErrors(hipMallocPitch((void**)&remoteHashed, &remoteHashedPitch,
		  	  	  	  MD_LENGTH*sizeof(u_char), dataPoolSize));
  checkCudaErrors(hipMalloc((void**)&remoteResult, sizeof(u_int)));
  checkCudaErrors(hipMemset(remoteResult, 0, sizeof(u_int)));
//  checkCudaErrors(hipMalloc((void**)&remoteScratch, sizeof(u_int)));
//  checkCudaErrors(hipMemset(remoteScratch, 0, sizeof(u_int)));
  if(times) {
    hipDeviceSynchronize();
   error = hipGetLastError();
    sdkStopTimer(&timer);
    times->mallocTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
    if(error!=hipSuccess)
    {
      fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
      exit(-1);
    }
  }
  
  //memcopies
  if(times) sdkStartTimer(&timer);
  checkCudaErrors(hipMemcpy2D(remoteData, remoteDataPitch, dataPool,
		  	  	  	  dataLength*sizeof(u_char), dataLength*sizeof(u_char),
		  	  	  	  dataPoolSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy2D(remotePool, remotePoolPitch, secretPool,
		  	  	  	  MD_LENGTH*sizeof(u_char), MD_LENGTH*sizeof(u_char),
		  	  	  	  sPoolSize, hipMemcpyHostToDevice));
  if(times){
    hipDeviceSynchronize();
    error = hipGetLastError();
    sdkStopTimer(&timer);
    times->memcpyHTDTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
    if(error!=hipSuccess)
    {
      fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
      exit(-1);
    }
  }
  
  //hash the data
  int threadsPerBlock = 512; // for 980ti 1024;
  int blocksPerThread = (dataPoolSize + threadsPerBlock +1) / threadsPerBlock;
  if(times) sdkStartTimer(&timer);
  DHashData<<<threadsPerBlock, blocksPerThread>>>(remoteData, remoteDataPitch, remoteHashed,
		  remoteHashedPitch, dataLength*sizeof(u_char), dataPoolSize);
  if(times){
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    times->hashTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
  }
  
  //compare the hashes against the pool
  int threadsPerBlock2 = 512;
  int blocksPerGrid2 = ((sPoolSize * dataPoolSize) + threadsPerBlock +1)/threadsPerBlock;
  if(times) sdkStartTimer(&timer);
  findMatch<<<blocksPerGrid2, threadsPerBlock2>>>(remoteHashed,
		  remoteHashedPitch, dataPoolSize, remotePool, remotePoolPitch,
		  sPoolSize, MD_LENGTH, remoteResult);
  if(times) {
    hipDeviceSynchronize();
    error = hipGetLastError();
    sdkStopTimer(&timer);
    times->findTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
    if(error!=hipSuccess)
    {
      fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
      exit(-1);
    } 
  }

  //return the int (bit array?)
  if(times) sdkStartTimer(&timer);
  checkCudaErrors(hipMemcpy(ret, remoteResult, sizeof(u_int),
		  hipMemcpyDeviceToHost));
  if(times){
    hipDeviceSynchronize();
    error = hipGetLastError();
    sdkStopTimer(&timer);
    times->memcpyDTHTime = sdkGetTimerValue(&timer);
    times->totalTime = times->mallocTime + times->hashTime + times->findTime +
    				   times->memcpyDTHTime + times->memcpyHTDTime;
    if(error!=hipSuccess)
    {
      fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
      exit(-1);
    }
  }
  //free and clean up memory
  hipFree(remoteData);
  hipFree(remoteHashed);
  hipFree(remotePool);
  hipFree(remoteResult);
//  hipFree(remoteScratch);
  sdkDeleteTimer(&timer);
  return chk;
}

__host__ __device__ int
getNthIdentifier(u_int* buff, int nth, int bValue, int length){
	u_int *temp = buff;
	u_int bytesPerInt = 4;
	u_int bitsPerByte = 8;
	u_int maxBit = length*bytesPerInt*bitsPerByte; //maximum number of bits that could be in the buff
	u_int maxNumberIdenties = maxBit / bValue;
	int nthStartBit = nth * bValue;
	u_int nthEndBit = nthStartBit + bValue - 1;
	u_int startIntIndex = nthStartBit / 31; //this will be the unsigned
								   //int which will contain our first bit.
	u_int startIntRangeBeg = startIntIndex * 32;
	u_int accum = 0;
	u_int exp = bValue -1;
	u_int startIntBitIndex = nthStartBit - startIntRangeBeg;
	if(nth > maxNumberIdenties || nth < 0 ||
			nthEndBit > maxBit ||
			nthStartBit < 0 ||
			bValue <= 0){ return -1;}
	//get the int closest to the first bit we need
	//if need get the rest of the bits
	for(int i = exp; i >= 0; i--){
		accum += ((temp[startIntIndex] >> (31 - startIntBitIndex)) & 1) << i;
		//check if still in range
		//iterate to next bit
		if((startIntBitIndex + 1) > 31){
		  startIntIndex++;
		  startIntBitIndex = 0;
		} else {
		  startIntBitIndex++;
		}
	}
	return accum;
}

extern "C" bool
handleDataV2(u_char* dataPool, u_int dataLength, u_int dataPoolSize,
  u_int* secretPool, u_int sPoolSize, u_int bValue, u_int maxIDs, u_int throwAway, u_int* ret, struct btimes* times)
{
  bool chk = true;
  u_char* remoteData;
  u_char* remoteHashed;
  u_int* remotePool;
  u_int* remoteResult;
  u_int* scratchResult;
  size_t remoteDataPitch = 0;
  size_t remoteHashedPitch = 0;
  StopWatchInterface* timer = NULL;
  hipError_t error;
  sdkCreateTimer(&timer);
  
  //mallocs
  if(times) sdkStartTimer(&timer); 
  checkCudaErrors(hipMallocPitch((void**)&remoteData, &remoteDataPitch,
		  	  	  	  dataLength*sizeof(u_char), dataPoolSize));
  checkCudaErrors(hipMallocPitch((void**)&remoteHashed, &remoteHashedPitch,
		  	  	  	  MD_LENGTH*sizeof(u_char), dataPoolSize));
  checkCudaErrors(hipMalloc((void**)&remotePool, sizeof(u_int)*sPoolSize));
  checkCudaErrors(hipMalloc((void**)&remoteResult,sizeof(u_int)*dataPoolSize));
  checkCudaErrors(hipMemset(remoteResult, 0, sizeof(u_int)*dataPoolSize));
  scratchResult = new u_int[dataPoolSize];
  if(times) {
    hipDeviceSynchronize();
   error = hipGetLastError();
    sdkStopTimer(&timer);
    times->mallocTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
    if(error!=hipSuccess)
    {
      fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
      exit(-1);
    }
  }
  
  //memcopies
  if(times) sdkStartTimer(&timer);
  checkCudaErrors(hipMemcpy2D(remoteData, remoteDataPitch, dataPool,
		  	  	  	  dataLength*sizeof(u_char), dataLength*sizeof(u_char),
		  	  	  	  dataPoolSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(remotePool, secretPool, sizeof(int)*sPoolSize, hipMemcpyHostToDevice));
  if(times){
    hipDeviceSynchronize();
    error = hipGetLastError();
    sdkStopTimer(&timer);
    times->memcpyHTDTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
    if(error!=hipSuccess)
    {
      fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
      exit(-1);
    }
  }
  
  //hash the data
  int threadsPerBlock = 512;
  int blocksPerThread = (dataPoolSize + threadsPerBlock -1) / threadsPerBlock;
//  printf("bValue: %d tValue: %d maxIDs: %d, sPoolSize: %d\n", bValue, throwAway, maxIDs, sPoolSize);
/*  for(int p = 0; p < sPoolSize; p++){
    printf("[%d]", secretPool[p]);
  }
  printf("\n");*/
  if(times) sdkStartTimer(&timer);
  DHashData<<<blocksPerThread, threadsPerBlock>>>(remoteData, remoteDataPitch, remoteHashed,
		  remoteHashedPitch, dataLength*sizeof(u_char), dataPoolSize);
  if(times){
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    times->hashTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
  }
  
  //compare the hashes against the pool
  int threadsPerBlock2 = 1024;
  int blocksPerGrid2 = ((dataPoolSize * (maxIDs - throwAway)) + threadsPerBlock -1)/threadsPerBlock;
  if(times) sdkStartTimer(&timer);
//  printf("check value before launch\n");
/*  for(int c = 0; c < sPoolSize; c++){                                        
/*
  printf("check value before launch\n");
  for(int c = 0; c < sPoolSize; c++){                                        
    bitset<32>t1(secretPool[c]);                                              
    printf("|%s_%d|", t1.to_string().c_str(),secretPool[c]);                  
  }   
  printf("\n"); 
  */ 
  findMatchV2<<<blocksPerGrid2, threadsPerBlock2>>>(remoteHashed,
		  remoteHashedPitch, dataPoolSize, remotePool, sPoolSize, bValue, maxIDs, throwAway, remoteResult);
  if(times) {
    hipDeviceSynchronize();
    error = hipGetLastError();
    sdkStopTimer(&timer);
    times->findTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
    if(error!=hipSuccess)
    {
      fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
      exit(-1);
    } 
  }

  //return the int (bit array?)
  if(times) sdkStartTimer(&timer);
  checkCudaErrors(hipMemcpy(scratchResult, remoteResult,dataPoolSize,
		  hipMemcpyDeviceToHost));
  int succMessCount = 0;
  for(int x=0; x <= dataPoolSize; x++){
    succMessCount += (scratchResult[x])?0:1;
  }
  if(times){
    hipDeviceSynchronize();
    error = hipGetLastError();
    sdkStopTimer(&timer);
    times->memcpyDTHTime = sdkGetTimerValue(&timer);
    times->totalTime = times->mallocTime + times->hashTime + times->findTime +
    				   times->memcpyDTHTime + times->memcpyHTDTime;
    if(error!=hipSuccess)
    {
      fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
      exit(-1);
    }
  }
  //free and clean up memory
  delete[] scratchResult;
  hipFree(remoteData);
  hipFree(remoteHashed);
  hipFree(remotePool);
  hipFree(remoteResult);
  sdkDeleteTimer(&timer);
  return chk;
}

extern "C" bool
handleDataV3(u_int seed, u_int numThreads, u_int* secretPool, u_int sPoolSize, u_int bValue, u_int maxIDs, u_int throwAway, u_int* ret, struct btimes* times)
{
  bool chk = true;
  bool broken = false;
  int series = 0;
  u_char* remoteHashed;
  u_int* remotePool;
  u_int* remoteResult;
  u_int* scratchResult;
  size_t remoteHashedPitch = 0;
  StopWatchInterface* timer = NULL;
  hipError_t error;
  sdkCreateTimer(&timer);

  while(broken == false){
    //mallocs
    if(times) sdkStartTimer(&timer); 
    checkCudaErrors(hipMallocPitch((void**)&remoteHashed, &remoteHashedPitch, MD_LENGTH*sizeof(u_char), numThreads));
    checkCudaErrors(hipMalloc((void**)&remotePool, sizeof(u_int)*sPoolSize));
    checkCudaErrors(hipMalloc((void**)&remoteResult,sizeof(u_int)*numThreads));
    checkCudaErrors(hipMemset(remoteResult, 0, sizeof(u_int)*numThreads));
    scratchResult = new u_int[numThreads];
    if(times) {
      hipDeviceSynchronize();
      error = hipGetLastError();
      sdkStopTimer(&timer);
      times->mallocTime = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      if(error!=hipSuccess)
      {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
      }
    }

    //copy
    if(times) sdkStartTimer(&timer);
    checkCudaErrors(hipMemcpy(remotePool, secretPool, sizeof(int)*sPoolSize, hipMemcpyHostToDevice));
    if(times){
      hipDeviceSynchronize();
      error = hipGetLastError();
      sdkStopTimer(&timer);
      times->memcpyHTDTime = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      if(error!=hipSuccess)
      {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
      }
    }

    int threadsPerBlock = 512;
    int blocksPerThread = (numThreads + threadsPerBlock -1) / threadsPerBlock;
    if(times) sdkStartTimer(&timer);
    hashSeed<<<blocksPerThread, threadsPerBlock>>>(seed, series, remoteHashed,
        remoteHashedPitch, numThreads*sizeof(u_char), numThreads);
    if(times){
      hipDeviceSynchronize();
      sdkStopTimer(&timer);
      times->hashTime = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
    }

    //compare the hashes against the pool
    int threadsPerBlock2 = 1024;
    int blocksPerGrid2 = ((numThreads * (maxIDs - throwAway)) + threadsPerBlock -1)/threadsPerBlock;
    if(times) sdkStartTimer(&timer);
    findMatchV2<<<blocksPerGrid2, threadsPerBlock2>>>(remoteHashed,
        remoteHashedPitch, numThreads, remotePool, sPoolSize, bValue, maxIDs, throwAway, remoteResult);
    if(times) {
      hipDeviceSynchronize();
      error = hipGetLastError();
      sdkStopTimer(&timer);
      times->findTime = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      if(error!=hipSuccess)
      {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
      }
    }
    //return the int (bit array?)
    if(times) sdkStartTimer(&timer);
    checkCudaErrors(hipMemcpy(scratchResult, remoteResult,numThreads*sizeof(u_int), hipMemcpyDeviceToHost));
    printf("what\n");
    int succMessCount = 0;
    for(int x=0; x <= numThreads; x++){
      printf("%d %d\n", x,scratchResult[x]);
      succMessCount += (scratchResult[x])?0:1;
    }
    printf("-\n");
    if(times){
      hipDeviceSynchronize();
      error = hipGetLastError();
      sdkStopTimer(&timer);
      times->memcpyDTHTime = sdkGetTimerValue(&timer);
      times->totalTime = times->mallocTime + times->hashTime + times->findTime +
        times->memcpyDTHTime + times->memcpyHTDTime;
      if(error!=hipSuccess)
      {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
      }
    }
    if(succMessCount > 0){ 
      fprintf(stderr, "broken\n");
      broken = true;
    }
    series++;
    delete[] scratchResult;
    hipFree(remoteHashed);
    hipFree(remotePool);
    hipFree(remoteResult);
    sdkDeleteTimer(&timer);


  }
  //free
  //return
  return chk;
}


extern "C" bool
sha1Kernel2D(u_char* src, int pWidth, int pHeight, u_char* dst,
	struct btimes* times)
{
  bool ret = true;
  size_t pitchSrc = 0;
  size_t pitchDst = 0;
  size_t widthSrc = pWidth * sizeof(u_char);
  size_t widthDst = MD_LENGTH * sizeof(u_char);
  size_t height = pHeight;
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  u_char* remoteSrc = NULL;
  u_char* remoteDst = NULL;  


/*
  sdkStartTimer(&timer);
  checkCudaErrors(hipMalloc((void**) &remoteSrc,  widthSrc * height));
  checkCudaErrors(hipMalloc((void**) &remoteDst, widthDst * height));
  checkCudaErrors(hipMemcpy(remoteSrc, src, widthSrc * height, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(remoteDst, src, widthDst * height, hipMemcpyHostToDevice));
  sdkStopTimer(&timer);
 // printf("malloc and copy [%f ms]\n", sdkGetTimerValue(&timer));
  sdkResetTimer(&timer);
*/

  //2d array implementation
  if(times) sdkStartTimer(&timer);
  checkCudaErrors(hipMallocPitch((void**)&remoteSrc,&pitchSrc, widthSrc, height));
  checkCudaErrors(hipMallocPitch((void**)&remoteDst,&pitchDst, widthDst, height));
  if(times){
    sdkStopTimer(&timer);
    times->mallocTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
  }

  //memcpy
  if(times) sdkStartTimer(&timer);
  checkCudaErrors(hipMemcpy2D(remoteSrc, pitchSrc, src, widthSrc * sizeof(u_char), 
                                widthSrc * sizeof(u_char), height, hipMemcpyHostToDevice));
  if(times){
    sdkStopTimer(&timer);
    times->memcpyHTDTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
  }
  
  //kernel launch, hash data

  int threadsPerBlock = 128;
  int blocksPerGrid = (pHeight + threadsPerBlock -1) / threadsPerBlock;
  if(times) sdkStartTimer(&timer);
  //  flattenHashData<<<height, 1>>>(remoteSrc, widthSrc, remoteDst, widthDst, height);
  DHashData<<<blocksPerGrid, threadsPerBlock>>>(remoteSrc, pitchSrc, remoteDst, pitchDst, widthSrc, height);
  if(times){
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    times->hashTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
  } 
 
  //copy data back from host 
  if(times) sdkStartTimer(&timer);
  checkCudaErrors(hipMemcpy2D(dst, widthDst * sizeof(u_char), remoteDst, pitchDst,
                                widthDst * sizeof(u_char), height, hipMemcpyDeviceToHost));
  if(times){
    hipDeviceSynchronize();
    sdkStopTimer(&timer); 
    times->memcpyDTHTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
  }

  checkCudaErrors(hipFree(remoteSrc));
  checkCudaErrors(hipFree(remoteDst));
  sdkDeleteTimer(&timer);
  return ret;
}

extern "C" bool
sha1KernelPerSecond(u_char* src, int pWidth, int pHeight, u_char* dst,
	struct btimes* times, u_int * hashes)
{
  bool ret = true;
  size_t pitchSrc = 0;
  size_t pitchDst = 0;
  size_t widthSrc = pWidth * sizeof(u_char);
  size_t widthDst = MD_LENGTH * sizeof(u_char);
  size_t height = pHeight;
  int repeats = 5000;
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  u_char* remoteSrc = NULL;
  u_char* remoteDst = NULL;  

  //2d array implementation
  checkCudaErrors(hipMallocPitch((void**)&remoteSrc,&pitchSrc, widthSrc, height));
  checkCudaErrors(hipMallocPitch((void**)&remoteDst,&pitchDst, widthDst, height));

  //memcpy
  checkCudaErrors(hipMemcpy2D(remoteSrc, pitchSrc, src, widthSrc * sizeof(u_char), 
                                widthSrc * sizeof(u_char), height, hipMemcpyHostToDevice));
  
  //kernel launch, hash data
  int threadsPerBlock = 128;
  int blocksPerGrid = (pHeight + threadsPerBlock -1) / threadsPerBlock;
  if(times) sdkStartTimer(&timer);
  //  flattenHashData<<<height, 1>>>(remoteSrc, widthSrc, remoteDst, widthDst, height);
  for(int x = 0; x < repeats; x++){
    DHashData<<<blocksPerGrid, threadsPerBlock>>>(remoteSrc, pitchSrc, remoteDst, pitchDst, widthSrc, height);
    *hashes += height;
  }
  if(times){
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    times->totalTime = (sdkGetTimerValue(&timer));
    sdkResetTimer(&timer);
  } 
 
  //copy data back from host 
  checkCudaErrors(hipMemcpy2D(dst, widthDst * sizeof(u_char), remoteDst, pitchDst,
                                widthDst * sizeof(u_char), height, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(remoteSrc));
  checkCudaErrors(hipFree(remoteDst));
  sdkDeleteTimer(&timer);
  return ret;
}

extern "C" bool
hashFindPerSecond(u_int* hashes, struct btimes* bench){
  int target = 45056;
  int maxRange = 1000;
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  u_char* data = new u_char[target * sizeof(int)];
  memset(data, 0, (target* sizeof(int)));
  u_char* check = new u_char[target * MD_LENGTH];
  memset(check, 0, (target*MD_LENGTH*sizeof(u_char)));
  u_char* out = new u_char[target * MD_LENGTH];
  memset(out, 0, (target*MD_LENGTH*sizeof(u_char)));
  int* temp = (int*)data;
  u_int bValue = 4;
  u_int maxIds = (1 << bValue) - 1;
  u_int identitiesPerMD = (INTS_PER_MD * BITS_PER_INT) / bValue;
  u_int throwAway = 0;
  for(int i = 0; i < target; i++){
    int ranNum = 1 + (rand() % maxRange);
    temp[i] = ranNum;
  }

  //hash all the random values into a checking array;
  for(int i = 0; i < target; i++){
    SHA1(data + (i*sizeof(int)), sizeof(int), check + (i*MD_LENGTH));
  }

  int checkSize = (maxIds / 32);
  u_int* pool = new u_int[checkSize];
  memset(pool, 0, checkSize*sizeof(u_int)); 
  u_char* currMD = NULL;
  for(int i = 0; i< (target); i++){
    currMD = check + (i + MD_LENGTH);
    for(int j = 0; j < identitiesPerMD; j++){
      int x = getNthIdentifier((u_int*) currMD, j, bValue, MD_LENGTH);
      setBit(pool, x);
    }
  }
  //hash the random values on the GPU
  handleDataPerSecond(data, (u_int) sizeof(int), (u_int) target, (u_int*) pool, 
      (u_int) target, bValue, maxIds, throwAway, (u_int*) hashes, bench); 
  delete[] data;
  delete[] out;
  delete[] check;
  sdkDeleteTimer(&timer);;
  return true;
}

extern "C" bool
hashesPerSecond(u_int* hashes,struct btimes* bench){
  int target = 45056/2;
  int maxRange = 1000;
  size_t height = target;
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  u_char* data = new u_char[target * sizeof(int)];
  memset(data, 0, (target* sizeof(int)));
  u_char* check = new u_char[target * MD_LENGTH];
  memset(check, 0, (target*MD_LENGTH*sizeof(u_char)));
  u_char* out = new u_char[target * MD_LENGTH];
  memset(out, 0, (target*MD_LENGTH*sizeof(u_char)));
  int* temp = (int*)data;

  for(int i = 0; i < target; i++){
    int ranNum = 1 + (rand() % maxRange);
    temp[i] = ranNum;
  }

  //hash all the random values into a checking array;
  for(int i = 0; i < target; i++){
    SHA1(data + (i*sizeof(int)), sizeof(int), check + (i*MD_LENGTH));
  }

  //hash the random values on the GPU
  sha1KernelPerSecond(data, sizeof(int), height, out, bench, hashes);
  
  delete[] data;
  delete[] out;
  delete[] check;
  sdkDeleteTimer(&timer);;
  return true;
}

extern "C" bool
handleDataPerSecond(u_char* dataPool, u_int dataLength, u_int dataPoolSize,
  u_int* secretPool, u_int sPoolSize, u_int bValue, u_int maxIDs, u_int throwAway, u_int* ret,struct btimes* times)
{
  bool chk = true;
  u_char* remoteData;
  u_char* remoteHashed;
  u_int* remotePool;
  u_int* remoteResult;
  u_int* scratchResult;
  size_t remoteDataPitch = 0;
  size_t remoteHashedPitch = 0;
  StopWatchInterface* timer = NULL;
  hipError_t error;
  int repeats = 5000;
  sdkCreateTimer(&timer);
  
  //mallocs
  checkCudaErrors(hipMallocPitch((void**)&remoteData, &remoteDataPitch,
		  	  	  	  dataLength*sizeof(u_char), dataPoolSize));
  checkCudaErrors(hipMallocPitch((void**)&remoteHashed, &remoteHashedPitch,
		  	  	  	  MD_LENGTH*sizeof(u_char), dataPoolSize));
  checkCudaErrors(hipMalloc((void**)&remotePool, sizeof(u_int)*sPoolSize));
  checkCudaErrors(hipMalloc((void**)&remoteResult, sizeof(u_int)*(maxIDs+1)));
  checkCudaErrors(hipMemset(remoteResult, 0, sizeof(u_int)*(maxIDs+1)));
  scratchResult = new u_int[maxIDs+1];

  //memcopies
  checkCudaErrors(hipMemcpy2D(remoteData, remoteDataPitch, dataPool,
		  	  	  	  dataLength*sizeof(u_char), dataLength*sizeof(u_char),
		  	  	  	  dataPoolSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(remotePool, secretPool, sizeof(int)*sPoolSize, hipMemcpyHostToDevice));
  
  if(times) sdkStartTimer(&timer);
  for(int x  = 0; x < repeats; x++){
    int threadsPerBlock = 128; //1024; for the 980ti
    int blocksPerThread = (dataPoolSize + threadsPerBlock -1) / threadsPerBlock;
    DHashData<<<blocksPerThread, threadsPerBlock>>>(remoteData, remoteDataPitch, remoteHashed,
		  remoteHashedPitch, dataLength*sizeof(u_char), dataPoolSize); 
    hipDeviceSynchronize();
    //compare the hashes against the pool
    int threadsPerBlock2 =  128;
    int blocksPerGrid2 = ((dataPoolSize * (maxIDs - throwAway)) + threadsPerBlock -1)/threadsPerBlock;
    findMatchV2<<<blocksPerGrid2, threadsPerBlock2>>>(remoteHashed,
		  remoteHashedPitch, dataPoolSize, remotePool, sPoolSize, bValue, maxIDs, throwAway, remoteResult);
    hipDeviceSynchronize();
    *ret += dataPoolSize;
  }
  if(times) {
    hipDeviceSynchronize();
    error = hipGetLastError();
    sdkStopTimer(&timer);
    times->totalTime = sdkGetTimerValue(&timer);
    sdkResetTimer(&timer);
    if(error!=hipSuccess)
    {
      fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
      exit(-1);
    } 
  }

  //return the int (bit array?)
  checkCudaErrors(hipMemcpy(scratchResult, remoteResult,sizeof(int)*(maxIDs+1),
		  hipMemcpyDeviceToHost));
  
  //free and clean up memory
  delete[] scratchResult;
  hipFree(remoteData);
  hipFree(remoteHashed);
  hipFree(remotePool);
  hipFree(remoteResult);
  sdkDeleteTimer(&timer);
  return chk;
}

extern "C" bool
sha1Kernel(u_char* src, int len, u_char* dest){
  bool ret = false;
  size_t size = len* sizeof(u_char);
  u_char* remoteSrc = NULL;
  u_char* remoteDst = NULL; 
  
  checkCudaErrors(hipMalloc((void**)&remoteSrc, size));
  checkCudaErrors(hipMalloc((void**)&remoteDst, MD_LENGTH*sizeof(u_char)));
  checkCudaErrors(hipMemcpy(remoteSrc, src, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(remoteDst, dest, MD_LENGTH*sizeof(u_char), hipMemcpyHostToDevice));

  singleSHA1<<<1,1>>>(remoteSrc, size, remoteDst);
  checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMemcpy(dest, remoteDst, 20*sizeof(u_char),
		  hipMemcpyDeviceToHost));
  

  //check against openssl's sha1

  u_char tempHash[20];
  SHA1(src, len, tempHash);
  
  checkCudaErrors(hipFree(remoteSrc));
  checkCudaErrors(hipFree(remoteDst));
  ret = (0 == memcmp(tempHash, dest, 20));
  return ret;
  
}



__host__ __device__ void
setBit(u_int* x, u_int val){
  u_int index = val / 32;
  u_int shift = 31 - (val % 32);
  x[index] |= 1 << shift;
}

__host__ __device__ void
clearBit(u_int* x, u_int val){
  u_int index = val / 32;
  u_int shift = 31 - (val % 32);
  x[index] &= ~(1 << shift);
}

__host__ __device__ void
toggleBit(u_int* x, u_int val){
  u_int index = val / 32;
  u_int shift = 31 - (val % 32);
  x[index] ^= 1 << shift;
}

__host__ __device__ int
isSet(u_int* x, u_int val){
  u_int index = val / 32;
  u_int shift = 31 - (val % 32);
  return (x[index] >> shift) & 1;
}

extern "C" bool 
vectorAdditionExample(const int argc, const char **argv, float *arrA, float *arrB, float *output, int len){
  bool ret = true;
  size_t size = len * sizeof(float);
  float* remoteA = NULL;
  float* remoteB = NULL;
  float* remoteOutput = NULL;

  //Create memory on the remote device
#ifdef DEBUG  
  printf("Allocate memory on the remote device\n");
#endif
  checkCudaErrors(hipMalloc((void**)&remoteA, size));
  checkCudaErrors(hipMalloc((void**)&remoteB, size));
  checkCudaErrors(hipMalloc((void**)&remoteOutput, size));

  //Copy data to remote device
#ifdef DEBUG
  printf("Copy memory from host to remote device\n"); 
#endif
  checkCudaErrors(hipMemcpy(remoteA, arrA, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(remoteB, arrB, size, hipMemcpyHostToDevice));

  //Launch Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
#ifdef DEBUG  
  printf("Launch Kernel\n");
#endif
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(remoteA, remoteB, remoteOutput, size);
  checkCudaErrors(hipGetLastError());

  //Copy output from remote device
#ifdef DEBUG 
  printf("Copy output data from the CUDA device to the host memory\n"); 
#endif
  checkCudaErrors(hipMemcpy(output, remoteOutput, size, hipMemcpyDeviceToHost));

  for(int i = 0; i < len; i++){
    if(fabs(arrA[i] + arrB[i] - output[i]) > 1e-5){
      fprintf(stderr, "Result verification failed at a element %d\n", i);
      ret = false;
    }
  }

  //Free the remote data 
  checkCudaErrors(hipFree(remoteA));
  checkCudaErrors(hipFree(remoteB));
  checkCudaErrors(hipFree(remoteOutput));

  return ret;	
}

/*
 * SHA-1 CPU implementation
 */
__device__ const unsigned char sha1_padding[64] =
{
        0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
};


/*
 * Prepare SHA-1 for execution.
 */
__host__ __device__ void 
sha1Init(unsigned long* total, unsigned long* state, unsigned char* buff)
{
        total[0] = 0;
        total[1] = 0;
        state[0] = 0x67452301;
        state[1] = 0xEFCDAB89;
        state[2] = 0x98BADCFE;
        state[3] = 0x10325476;
        state[4] = 0xC3D2E1F0;
}


/*
 * Process one block of data.
 */
__host__ __device__ void 

sha1ProcessBlock(unsigned long* total, unsigned long* state, unsigned char* buff, unsigned char data[64])
{
        unsigned long temp, W[16]={0,}, A, B, C, D, E;
  
        GET_UINT32_BE( W[ 0], data,  0 );
        GET_UINT32_BE( W[ 1], data,  4 );
        GET_UINT32_BE( W[ 2], data,  8 );
        GET_UINT32_BE( W[ 3], data, 12 );
        GET_UINT32_BE( W[ 4], data, 16 );
        GET_UINT32_BE( W[ 5], data, 20 );
        GET_UINT32_BE( W[ 6], data, 24 );
        GET_UINT32_BE( W[ 7], data, 28 );
        GET_UINT32_BE( W[ 8], data, 32 );
        GET_UINT32_BE( W[ 9], data, 36 );
        GET_UINT32_BE( W[10], data, 40 );
        GET_UINT32_BE( W[11], data, 44 );
        GET_UINT32_BE( W[12], data, 48 );
        GET_UINT32_BE( W[13], data, 52 );
        GET_UINT32_BE( W[14], data, 56 );
        GET_UINT32_BE( W[15], data, 60 );
  
#define S(x,n) ((x << n) | ((x & 0xFFFFFFFF) >> (32 - n)))

#define R(t)                                            \
(                                                       \
    temp = W[(t -  3) & 0x0F] ^ W[(t - 8) & 0x0F] ^     \
           W[(t - 14) & 0x0F] ^ W[ t      & 0x0F],      \
    ( W[t & 0x0F] = S(temp,1) )                         \
)

#define P(a,b,c,d,e,x)                                  \
{                                                       \
    e += S(a,5) + F(b,c,d) + K + x; b = S(b,30);        \
}

        A = state[0];
        B = state[1];
        C = state[2];
        D = state[3];
        E = state[4];
  
#define F(x,y,z) (z ^ (x & (y ^ z)))
#define K 0x5A827999
  
        P( A, B, C, D, E, W[0]  );
        P( E, A, B, C, D, W[1]  );
        P( D, E, A, B, C, W[2]  );
        P( C, D, E, A, B, W[3]  );
        P( B, C, D, E, A, W[4]  );
        P( A, B, C, D, E, W[5]  );
        P( E, A, B, C, D, W[6]  );
        P( D, E, A, B, C, W[7]  );
        P( C, D, E, A, B, W[8]  );
        P( B, C, D, E, A, W[9]  );
        P( A, B, C, D, E, W[10] );
        P( E, A, B, C, D, W[11] );
        P( D, E, A, B, C, W[12] );
        P( C, D, E, A, B, W[13] );
        P( B, C, D, E, A, W[14] );
        P( A, B, C, D, E, W[15] );
        P( E, A, B, C, D, R(16) );
        P( D, E, A, B, C, R(17) );
        P( C, D, E, A, B, R(18) );
        P( B, C, D, E, A, R(19) );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0x6ED9EBA1
  
        P( A, B, C, D, E, R(20) );
        P( E, A, B, C, D, R(21) );
        P( D, E, A, B, C, R(22) );
        P( C, D, E, A, B, R(23) );
        P( B, C, D, E, A, R(24) );
        P( A, B, C, D, E, R(25) );
        P( E, A, B, C, D, R(26) );
        P( D, E, A, B, C, R(27) );
        P( C, D, E, A, B, R(28) );
        P( B, C, D, E, A, R(29) );
        P( A, B, C, D, E, R(30) );
        P( E, A, B, C, D, R(31) );
        P( D, E, A, B, C, R(32) );
        P( C, D, E, A, B, R(33) );
        P( B, C, D, E, A, R(34) );
        P( A, B, C, D, E, R(35) );
        P( E, A, B, C, D, R(36) );
        P( D, E, A, B, C, R(37) );
        P( C, D, E, A, B, R(38) );
        P( B, C, D, E, A, R(39) );

#undef K
#undef F

#define F(x,y,z) ((x & y) | (z & (x | y)))
#define K 0x8F1BBCDC

        P( A, B, C, D, E, R(40) );
        P( E, A, B, C, D, R(41) );
        P( D, E, A, B, C, R(42) );
        P( C, D, E, A, B, R(43) );
        P( B, C, D, E, A, R(44) );
        P( A, B, C, D, E, R(45) );
        P( E, A, B, C, D, R(46) );
        P( D, E, A, B, C, R(47) );
        P( C, D, E, A, B, R(48) );
        P( B, C, D, E, A, R(49) );
        P( A, B, C, D, E, R(50) );
        P( E, A, B, C, D, R(51) );
        P( D, E, A, B, C, R(52) );
        P( C, D, E, A, B, R(53) );
        P( B, C, D, E, A, R(54) );
        P( A, B, C, D, E, R(55) );
        P( E, A, B, C, D, R(56) );
        P( D, E, A, B, C, R(57) );
        P( C, D, E, A, B, R(58) );
        P( B, C, D, E, A, R(59) );

#undef K
#undef F

#define F(x,y,z) (x ^ y ^ z)
#define K 0xCA62C1D6
  
        P( A, B, C, D, E, R(60) );
        P( E, A, B, C, D, R(61) );
        P( D, E, A, B, C, R(62) );
        P( C, D, E, A, B, R(63) );
        P( B, C, D, E, A, R(64) );
        P( A, B, C, D, E, R(65) );
        P( E, A, B, C, D, R(66) );
        P( D, E, A, B, C, R(67) );
        P( C, D, E, A, B, R(68) );
        P( B, C, D, E, A, R(69) );
        P( A, B, C, D, E, R(70) );
        P( E, A, B, C, D, R(71) );
        P( D, E, A, B, C, R(72) );
        P( C, D, E, A, B, R(73) );
        P( B, C, D, E, A, R(74) );
        P( A, B, C, D, E, R(75) );
        P( E, A, B, C, D, R(76) );
        P( D, E, A, B, C, R(77) );
        P( C, D, E, A, B, R(78) );
        P( B, C, D, E, A, R(79) );

#undef K
#undef F

        state[0] += A;
        state[1] += B;
        state[2] += C;
        state[3] += D;
        state[4] += E;
}


/*
 * Splits input message into blocks and processes them one by one. Also
 * checks how many 0 need to be padded and processes the last, padded, block.
 */
__host__ __device__ void 
sha1Update(unsigned long* total, unsigned long* state, unsigned char* buff, unsigned char *input, int ilen)
{
        int fill;
        unsigned long left;
  
        if ( ilen <= 0 )
                return;
  
        left = total[0] & 0x3F;
        fill = 64 - left;
  
        total[0] += ilen;
        total[0] &= 0xFFFFFFFF;

        if (total[0] < (unsigned long) ilen)
                total[1]++;
  
        if ( left && ilen >= fill ) {
                memcpy((void *) (buff + left), (void *) input, fill);
                sha1ProcessBlock(total, state, buff, buff);
                input += fill;
                ilen  -= fill;
                left = 0;
        }
  
        while ( ilen >= 64 ) {
                sha1ProcessBlock(total, state, buff, input);
                input += 64;
                ilen  -= 64;
        }
  
        if ( ilen > 0 ) {
                memcpy( (void *) (buff + left), (void *) input, ilen );
        }
}


/*
 * Process padded block and return hash to user.
 */
__host__ __device__ void 
sha1Finish(unsigned long* total,unsigned long* state, unsigned char* buff, unsigned char *output)
{

        unsigned long last, padn;
        unsigned long high, low;
        unsigned char msglen[8];


        high = (total[0] >> 29) | (total[1] <<  3);
        low  = (total[0] <<  3);

        PUT_UINT32_BE(high, msglen, 0);
        PUT_UINT32_BE(low,  msglen, 4);

        last = total[0] & 0x3F;
        padn = (last < 56 ) ? ( 56 - last ) : ( 120 - last);

        sha1Update(total, state, buff, (unsigned char *) sha1_padding, padn);
        sha1Update(total, state, buff, msglen, 8);

        PUT_UINT32_BE(state[0], output,  0);
        PUT_UINT32_BE(state[1], output,  4);
        PUT_UINT32_BE(state[2], output,  8);
        PUT_UINT32_BE(state[3], output, 12);
        PUT_UINT32_BE(state[4], output, 16);
}

/*
 * Execute SHA-1
 */

__host__ __device__ void
sha1Device(unsigned char *input, int ilen, unsigned char *output) {
        unsigned long total[2];
        unsigned long state[5];
        unsigned char buff[64];

        sha1Init( total, state, buff );
        sha1Update( total, state, buff, input, ilen );
        sha1Finish( total, state, buff, output );

}


string getCurrGpuTemp(){
  system("./updateCurrentGpuTemp");
  string ret;
  ifstream infile("./.currGpuTemp");
  if(infile.good()){
    getline(infile, ret);
  }
  return ret;
}
